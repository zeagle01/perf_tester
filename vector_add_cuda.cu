#include "hip/hip_runtime.h"




#include ""
#include "hip/hip_runtime.h"
#include "vector_add_cuda.h"

__global__ void vector_add(int num,float* c, float* a, float* b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num)
	{
		c[tid] = a[tid] + b[tid];
	}
}

void Cuda_Vector_Add::init(int size)
{
	m_size = size;
	m_a.resize(size);
	m_b.resize(size);
	m_c.resize(size);

	int size_in_byte = size * sizeof(float);

	free_cuda();
	hipMalloc(&da, size_in_byte);
	hipMalloc(&db, size_in_byte);
	hipMalloc(&dc, size_in_byte);
	for (int i = 0; i < m_a.size(); i++)
	{
		m_a[i] = 1.f;
		m_b[i] = 1.f;
		m_c[i] = 0.f;
	}

	hipMemcpy(da, m_a.data(), size_in_byte, hipMemcpyHostToDevice);
	hipMemcpy(db, m_b.data(), size_in_byte, hipMemcpyHostToDevice);
	hipMemcpy(dc, m_c.data(), size_in_byte, hipMemcpyHostToDevice);
}

void Cuda_Vector_Add::run() 
{
	int tpb = 128;
	int bpg = (m_size - 1) / tpb + 1;
	vector_add << <bpg, tpb >> > (m_size, dc, da, db);
}

void Cuda_Vector_Add::sync_wait()
{
	hipDeviceSynchronize();
};

size_t Cuda_Vector_Add::get_size_in_byte()
{
	return m_size * sizeof(float);
}

bool Cuda_Vector_Add::verify() 
{


	hipMemcpy(m_a.data(), da, get_size_in_byte(), hipMemcpyDeviceToHost);
	hipMemcpy(m_b.data(), db, get_size_in_byte(), hipMemcpyDeviceToHost);
	hipMemcpy(m_c.data(), dc, get_size_in_byte(), hipMemcpyDeviceToHost);

	for (int i = 0; i < m_size; i++)
	{
		if (m_c[i] != m_a[i] + m_b[i])
		{
			return false;
			break;
		}
	}
	return true;
};

Cuda_Vector_Add::~Cuda_Vector_Add()
{
	free_cuda();
}

void Cuda_Vector_Add::free_cuda()
{
	if (da)
	{

		hipFree(da);
	}
	if (db)
	{

		hipFree(db);
	}
	if (dc)
	{
		hipFree(dc);
	}

}

/////////////////////////////////////////
__global__ void vector_add_multiply(int num, float* c, float* a, float* b,int compute_intensity)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num)
	{
		float cc = c[tid];
		for (int i = 0; i < compute_intensity; i++)
		{
			cc = cc * a[tid] + b[tid];
		}
		c[tid] = cc;
	}
}

void Cuda_Vector_Add_Multiply::init(int size)
{
	m_size = size;
	m_a.resize(size);
	m_b.resize(size);
	m_c.resize(size);

	int size_in_byte = size * sizeof(float);

	free_cuda();
	hipMalloc(&da, size_in_byte);
	hipMalloc(&db, size_in_byte);
	hipMalloc(&dc, size_in_byte);
	for (int i = 0; i < m_a.size(); i++)
	{
		m_a[i] = 0.9f;
		m_b[i] = 1.f;
		m_c[i] = 0.f;
	}

	hipMemcpy(da, m_a.data(), size_in_byte, hipMemcpyHostToDevice);
	hipMemcpy(db, m_b.data(), size_in_byte, hipMemcpyHostToDevice);
	hipMemcpy(dc, m_c.data(), size_in_byte, hipMemcpyHostToDevice);
}

void Cuda_Vector_Add_Multiply::run() 
{
	int tpb = 128;
	int bpg = (m_size - 1) / tpb + 1;
	vector_add_multiply << <bpg, tpb >> > (m_size, dc, da, db, m_compute_intensity);
}

void Cuda_Vector_Add_Multiply::sync_wait()
{
	hipDeviceSynchronize();
};

size_t Cuda_Vector_Add_Multiply::get_size_in_byte()
{
	return m_size * sizeof(float) * m_compute_intensity;
}

bool Cuda_Vector_Add_Multiply::verify() 
{


	auto data_in_byte = m_size * sizeof(float);
	hipMemcpy(m_a.data(), da, data_in_byte, hipMemcpyDeviceToHost);
	hipMemcpy(m_b.data(), db, data_in_byte, hipMemcpyDeviceToHost);
	hipMemcpy(m_c.data(), dc, data_in_byte, hipMemcpyDeviceToHost);

	for (int i = 0; i < m_size; i++)
	{
		if (std::abs(m_c[i] - expect_value) > 1e-3f)
		{
			return false;
			break;
		}
	}
	return true;
};

Cuda_Vector_Add_Multiply::~Cuda_Vector_Add_Multiply()
{
	free_cuda();
}

void Cuda_Vector_Add_Multiply::free_cuda()
{
	if (da)
	{

		hipFree(da);
	}
	if (db)
	{

		hipFree(db);
	}
	if (dc)
	{
		hipFree(dc);
	}

}
