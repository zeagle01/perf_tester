#include "hip/hip_runtime.h"




#include ""
#include "hip/hip_runtime.h"
#include "vector_add_cuda.h"

__global__ void vector_add(int num,float* c, float* a, float* b)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < num)
	{
		c[tid] = a[tid] + b[tid];
	}
}

void Cuda_Vector_Add::init(int size)
{
	m_size = size;
	m_a.resize(size);
	m_b.resize(size);
	m_c.resize(size);

	int size_in_byte = size * sizeof(float);

	free_cuda();
	hipMalloc(&da, size_in_byte);
	hipMalloc(&db, size_in_byte);
	hipMalloc(&dc, size_in_byte);
	for (int i = 0; i < m_a.size(); i++)
	{
		m_a[i] = 1.f;
		m_b[i] = 1.f;
		m_c[i] = 0.f;
	}

	hipMemcpy(da, m_a.data(), size_in_byte, hipMemcpyHostToDevice);
	hipMemcpy(db, m_b.data(), size_in_byte, hipMemcpyHostToDevice);
	hipMemcpy(dc, m_c.data(), size_in_byte, hipMemcpyHostToDevice);
}

void Cuda_Vector_Add::run() 
{
	int tpb = 128;
	int bpg = (m_size - 1) / tpb + 1;
	vector_add<<<bpg,tpb>>>(m_size, dc, da, db);
}

void Cuda_Vector_Add::sync_wait()
{
	hipDeviceSynchronize();
};

size_t Cuda_Vector_Add::get_size_in_byte()
{
	return m_size * sizeof(float);
}

bool Cuda_Vector_Add::verify() 
{


	hipMemcpy(m_a.data(), da, get_size_in_byte(), hipMemcpyDeviceToHost);
	hipMemcpy(m_b.data(), db, get_size_in_byte(), hipMemcpyDeviceToHost);
	hipMemcpy(m_c.data(), dc, get_size_in_byte(), hipMemcpyDeviceToHost);

	for (int i = 0; i < m_size; i++)
	{
		if (m_c[i] != m_a[i] + m_b[i])
		{
			return false;
			break;
		}
	}
	return true;
};

Cuda_Vector_Add::~Cuda_Vector_Add()
{
	free_cuda();
}

void Cuda_Vector_Add::free_cuda()
{
	if (da)
	{

		hipFree(da);
	}
	if (db)
	{

		hipFree(db);
	}
	if (dc)
	{
		hipFree(dc);
	}

}
